
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdio>

#define N 3

__global__ void inc(int *a) {
  int i = blockIdx.x;
  if (i < N) {
    a[i]++;
  }
#ifdef __CUDA_ARCH__
  printf ("Hello World!  From device b.t %d.%d\n", i, threadIdx.x);
#endif
}

int main() {
  int ha[N], *da;
  hipMalloc((void **)&da, N*sizeof(int));
  for (int i = 0; i < N; ++i) {
    ha[i] = i;
  }
  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
  inc <<<N, 1>>>(da);
  hipMemcpy(ha, da, N*sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; ++i) {
    assert(ha[i] == i + 1);
  }
  hipFree(da);
  printf("All asserts pass - looks like cuda is working!\n");
  return 0;
}
